#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <string>
#include <dirent.h>
#include "utils.h"

#define BLOCK_SIZE 16

__global__ void sobelKernel(unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 1 && y >= 1 && x < width - 1 && y < height - 1) {
        int gx = -input[(y - 1) * width + (x - 1)] - 2 * input[y * width + (x - 1)] - input[(y + 1) * width + (x - 1)]
                 + input[(y - 1) * width + (x + 1)] + 2 * input[y * width + (x + 1)] + input[(y + 1) * width + (x + 1)];

        int gy = -input[(y - 1) * width + (x - 1)] - 2 * input[(y - 1) * width + x] - input[(y - 1) * width + (x + 1)]
                 + input[(y + 1) * width + (x - 1)] + 2 * input[(y + 1) * width + x] + input[(y + 1) * width + (x + 1)];

        output[y * width + x] = min(255, abs(gx) + abs(gy));
    }
}

void processImage(const std::string& inputPath, const std::string& outputPath) {
    int width, height;
    unsigned char* inputImage = readPGM(inputPath.c_str(), &width, &height);
    unsigned char* outputImage = new unsigned char[width * height];

    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, width * height);
    hipMalloc(&d_output, width * height);

    hipMemcpy(d_input, inputImage, width * height, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    sobelKernel<<<numBlocks, threadsPerBlock>>>(d_input, d_output, width, height);

    hipMemcpy(outputImage, d_output, width * height, hipMemcpyDeviceToHost);

    writePGM(outputPath.c_str(), outputImage, width, height);

    hipFree(d_input);
    hipFree(d_output);
    delete[] inputImage;
    delete[] outputImage;
}

int main() {
    std::vector<std::string> files = getPGMFiles("images/");
    for (const auto& file : files) {
        std::string inputPath = "images/" + file;
        std::string outputPath = "images/output_" + file;
        std::cout << "Processing " << inputPath << std::endl;
        processImage(inputPath, outputPath);
    }
    return 0;
}
